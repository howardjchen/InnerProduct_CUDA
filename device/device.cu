
#include <hip/hip_runtime.h>
#include <stdio.h> 

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);

    printf("Device name: %s\n", prop.name);
		printf("TotalGlobalMem: %d kB\n", prop.totalGlobalMem/1024);
		printf("SharedMemPerBlock: %d kB\n", prop.sharedMemPerBlock/1024);
		printf("MaxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
		printf("MaxThreadsDim [x]: %d [y]: %d [z]: %d\n", 
					prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("MaxGridSize [x]: %d [y]: %d [z]: %d\n", 
					prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("MultiProcessorCount: %d\n", prop.multiProcessorCount);
		printf("MaxThreadsPerMultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
  }
}
