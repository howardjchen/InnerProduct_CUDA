#include "hip/hip_runtime.h"
// This program executes a typical convolutional layer in regular CNNs
#include <iostream>
#include "cnnConvLayer.h"
using namespace std;

// This is the CPU version, please don't modify it
void convLayerCPU()
{
	// declarations for bunch of indexing parameters
	int fn, sli, fmy, fmx, y, x;
	int sum, ifmy, ifmx, ofmy, ofmx;
	int filtIdx, inNeuIdx, outNeuIdx, outIdx;
	int filtVol = FMDEPTH * FILTSIZE * FILTSIZE;
	int filtArea = FILTSIZE * FILTSIZE;
	int fmArea = FMSIZE *FMSIZE;
	int outArea = FMSIZE/2 * FMSIZE/2;

	// Convolution
	for(fn = 0; fn < FILTNUM; fn++){
		for(fmy = 0; fmy < FMSIZE; fmy += STRIDE){
			for(fmx = 0; fmx < FMSIZE; fmx += STRIDE){
				sum = 0;
				for(sli = 0; sli < FMDEPTH; sli++){
					for(y = 0; y < FILTSIZE; y++){
						for(x = 0; x < FILTSIZE; x++){
							ifmy = fmy - FILTSIZE / 2 + y;
							ifmx = fmx - FILTSIZE / 2 + x;
							filtIdx = fn*filtVol + sli*filtArea + y*FILTSIZE + x;
							inNeuIdx = sli*fmArea + ifmy*FMSIZE + ifmx;
							if(ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)
								sum += filt[filtIdx] * inNeu[inNeuIdx];
						}
					}
				}
				// Activation - ReLU
				outNeuIdx = fn*fmArea + fmy*FMSIZE + fmx;
				if(sum <= 0)
					outNeu[outNeuIdx] = 0;
				else
					outNeu[outNeuIdx] = sum;
			}
		}
	}

	// Max Pooling with Window Size 2x2
	int max, tmpVal;
	for(sli = 0; sli < FILTNUM; sli++){
		for(fmy = 0; fmy < FMSIZE/2 ; fmy += 1){
			for(fmx = 0; fmx < FMSIZE/2 ; fmx += 1){
				outNeuIdx = sli*fmArea + fmy*2*FMSIZE + fmx*2;
				max = outNeu[outNeuIdx];
				for(y = 0; y < 2; y++){
					for(x = 0; x < 2; x++){
						ofmy = fmy*2 + y;
						ofmx = fmx*2 + x;
						outNeuIdx = sli*fmArea + ofmy*FMSIZE + ofmx;
						tmpVal = outNeu[outNeuIdx];	
						if(tmpVal > max)
							max = tmpVal;
					}
				}
				outIdx = sli*outArea + fmy*FMSIZE/2 + fmx;
				outCPU[outIdx] = max;
			}
		}
	}
}

/***	Implement your CUDA Kernel here	***/
__global__
void convLayerGPU()
{
}
/***	Implement your CUDA Kernel here	***/

int main()
{
	int convLayerCPUExecTime, convLayerGPUExecTime;
	init();
		
	timespec time_begin, time_end;                                                 
  clock_gettime(CLOCK_REALTIME, &time_begin);

	convLayerCPU();

  clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerCPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "CPU time for executing a typical convolutional layer = " 
			 <<  convLayerCPUExecTime / 1000 << "ms" << endl;

  clock_gettime(CLOCK_REALTIME, &time_begin);
	/***	Lunch your CUDA Kernel here	***/

	convLayerGPU<<<1,1>>>(); // Lunch the kernel
	
	hipDeviceSynchronize(); // Do synchronization before clock_gettime()
	/***	Lunch your CUDA Kernel here	***/
  clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerGPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "GPU time for executing a typical convolutional layer = " 
			 << convLayerGPUExecTime / 1000 << "ms" << endl;

	if(checker()){
		cout << "Congratulations! You pass the check." << endl;
		cout << "Speedup: " << (float)convLayerCPUExecTime / convLayerGPUExecTime << endl;
	}
	else
		cout << "Sorry! Your result is wrong." << endl;

	ending();
	
	return 0;
}
